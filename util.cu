#include "hip/hip_runtime.h"
#include "util.h"
#include <stdbool.h>
#include <hip/hip_runtime.h>

int euclid_edgelen_h(int i, int j, int *coords);
int tour_len(int **tour, int *coords, int N)
{
	if(N == 0)
		return 0;
	int len = 0;
	for (int i=0;i<N;i++)
	{
		len += euclid_edgelen_h(*(*tour+i), *(*tour+(i+1)%N), coords);
	}
	return len;
}

__device__ void reverse_S(int array[], int from, int N)
{
	int head = from;
	int tail = N - 1;

	while(head < tail)
	{
		array[head] = array[tail] + array[head];
		array[tail] = array[head] - array[tail];
		array[head] = array[head] - array[tail];
		++head;
		--tail;
	}
}
__device__ void swap_S(int array[], int i, int j, int N)
{
	if (i == j)
		return;
	if(i<N && j<N)
	{
		array[i] = array[i] + array[j];
		array[j] = array[i] - array[j];
		array[i] = array[i] - array[j];
	}
	else
	{
		printf("swap fail: out of bound\n");
	}
}

int euclid_edgelen_h(int i, int j, int *coords) 
{
	int dist;
	float dx, dy;
	dx = coords[2 * i] - coords[2 * j];
	dy = coords[2 * i + 1] - coords[2 * j + 1];
	dist = (int)(sqrtf(dx*dx + dy*dy) + 0.5);
	return dist;
}


int read_euc2d_file(char *filename, int **coords, int *dimension) {
	FILE *fp;
	fp = fopen(filename, "r");
	if (fp == NULL) {
		printf("file read error (1)\n");
		exit(-1);
	}

	while (!feof(fp)) {
		char buffer[256];
		if(!fgets(buffer, 256, fp)) {
			if (!feof(fp)) {
				fprintf(stderr, "file read failure\n");
				exit(-1);
			}
		}
		if (strstr(buffer, "DIMENSION")) {
			sscanf(buffer, "DIMENSION : %d", dimension);
		}
	}
	rewind(fp);
	*coords = (int*)malloc(*dimension * 2 * sizeof(int));

	if (coords == NULL) {
		fprintf(stderr, "out of heap memory");
		exit(-1);
	}

	bool ncs = false;
	while (!feof(fp)) {
		char buffer[256];
		int i, x, y;
		if(!fgets(buffer, 256, fp)) {
			if (!feof(fp)) {
				fprintf(stderr, "file read failure\n");
				exit(-1);
			}
		}
       
		if (!ncs && !strstr(buffer, "NODE_COORD_SECTION"))
			continue;
		else if (!ncs && strstr(buffer, "NODE_COORD_SECTION")) {
			ncs = true;
			continue;
		}
		else {
			sscanf(buffer, "%d %d %d", &i, &x, &y);
			i--;
			(*coords)[i * 2] = x;
			(*coords)[i * 2 + 1] = y;
		}

	}
	return 0;
}
